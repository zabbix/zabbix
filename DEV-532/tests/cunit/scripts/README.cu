#include "hip/hip_runtime.h"
A lot of Zabbix components are highly integrated and hard to separate for unit
tests. Because of that the idea is to embedd unit tests in Zabbix binaries 
(server, proxy, agent) with a help of prepare script.

The embedding is done by including unit tests at the end of corresponding
source files and redefining the daemon entry functions with a test runner 
functions.

The unit tests are embedded in three target binaries - zabbix_server, 
zabbix_proxy and zabbix_agentd.

The unit test sources are kept in target specific directory (zabbix_server, 
zabbix_proxy, zabbix_agentd) and have the same directory structure as Zabbix
source files. 

When preparing Zabbix sources for unit tests the prepare script does the 
following:
 * scan the directories for unit tests
 * append/insert unit test source includes in corresponding Zabbix source files
 * copy the test runner sources template file from test directory to target
   source directory
 * generate test initialization and runner functions and append them to the 
   copied source templates
 * set LIBS/CFLAGS variables and run ./configure

Directory structure:
  ./zbxcunit.h               // the common include file for unit tests
  ./zabbix_server_cu.c       // server test runner template
  ./zabbix_proxy_cu.c        // proxy test runner template  
  ./zabbix_agent_cu.c        // server test  runner template

  ./zabbix_server/libs/zbxcomms/comms.c         // server comms unit tests
  ./zabbix_server/libs/zbxdbcache/valuecache.c  // server value cache unit tests

  ./zabbix_proxy/libs/zbxcomms/comms.c          // proxy comms unit tests

  ./zabbix_agent/zabbix_agent/logfiles.c        // agent log file unit tests

The test examples are based on CUnit, but any other C unit testing framework
could be used instead.


The prepare script forwards all command line parameters to configure except
the internal options:
  --revert          - revert the changes to the Zabbix source. Must be done 
                      before commiting any changes to svn.
  --skip-configure  - prepare the sources, but don't invoke configure
  --testsrc=<path>  - set unit test source directory
  --report=<path>   - set report output directory for automated tests
  --mode=<mode>     - basic - run tests and generate output in console
                    - automated - run tests and generate report files

